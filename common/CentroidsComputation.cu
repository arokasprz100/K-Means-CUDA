#include "CentroidsComputation.h"

#include <algorithm>
#include <random>
#include <numeric>


std::vector<unsigned> getRandomInitialCentroids(unsigned number_of_rows, unsigned number_of_centroids) {
    std::vector<unsigned> dataset_indexes(number_of_rows);
    std::iota(dataset_indexes.begin(), dataset_indexes.end(), 0);
    std::random_device rd{};
    std::mt19937 rng{rd()};
    std::shuffle(dataset_indexes.begin(), dataset_indexes.end(), rng);

    std::vector<unsigned> centroids;
    for(unsigned i = 0; i < number_of_centroids; ++i) {
        centroids.push_back(dataset_indexes.at(i));
    }
    return centroids;
}