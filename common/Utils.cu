#include "Utils.h"

hipError_t handleCudaErrors(hipError_t error_code) {
    if(error_code != hipSuccess) {
        std::cout << "CUDA-related error: " << hipGetErrorString(error_code) << std::endl;
        exit(EXIT_FAILURE);
    }
    return error_code;
}


unsigned getNumberOfStreamingMultiprocessors() {
    int deviceId = 0;
    handleCudaErrors( hipGetDevice(&deviceId) );
    hipDeviceProp_t deviceProp{};
    handleCudaErrors( hipGetDeviceProperties(&deviceProp, deviceId) );
    return static_cast<unsigned>(deviceProp.multiProcessorCount);
}


void printInformationAboutGPUDevice() {
    int deviceId = 0;
    handleCudaErrors( hipGetDevice(&deviceId) );
    hipDeviceProp_t deviceProp{};
    handleCudaErrors( hipGetDeviceProperties(&deviceProp, deviceId) );
    std::cout << "[DEVICE] Name: " << deviceProp.name << ", number of SMs: " << deviceProp.multiProcessorCount
              << ", shared memory per SM: " << deviceProp.sharedMemPerMultiprocessor << std::endl;
}